#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>
#include </usr/local/cuda/include/hip/hip_runtime.h>
#include <iostream>
#include <chrono>

using namespace cv;
using namespace std;
using namespace chrono;


__global__ void binarization_kernel(uchar*, uchar*, int, int, int);
__global__ void xor_kernel(uchar*, uchar*, uchar*, int, int);
__global__ void mark_differences_kernel(uchar*, uchar*, uchar*, int, int);

void printGPUinfo() {
    hipDeviceProp_t device;
    hipGetDeviceProperties(&device, 0);
    cout << "\n------------ GPU INFO ------------" << endl;
    cout << "GPU: " << device.name << endl;
    cout << "Max threads per block: " << device.maxThreadsPerBlock << endl;
}

/*********** MAIN ***********/
int main(int argc, char* argv[]) {
    if (argc < 5) {
        cerr << "Usage: " << argv[0] << " <img1> <img2> <threads_x> <threads_y>" << endl;
        return -1;
    }

    Mat img1 = imread(argv[1], IMREAD_COLOR);
    Mat img2 = imread(argv[2], IMREAD_GRAYSCALE);
    if (img1.empty() || img2.empty()) {
        cerr << "Error: Empty images!" << endl;
        return -1;
    }

    int th = 150;
    Mat display = img1.clone();
    cvtColor(img1, img1, COLOR_BGR2GRAY);

    dim3 NumThreadsPerBlock(atoi(argv[3]), atoi(argv[4])), nBlocks;

    nBlocks.x = img1.cols / NumThreadsPerBlock.x + ((img1.cols % NumThreadsPerBlock.x) == 0 ? 0 : 1);
    nBlocks.y = img1.rows / NumThreadsPerBlock.y + ((img1.rows % NumThreadsPerBlock.y) == 0 ? 0 : 1);
    
    printGPUinfo();
    
    
    size_t total_size = img1.rows * img1.cols * sizeof(uchar);
    size_t total_size_color = img1.rows * img1.cols * 3 * sizeof(uchar);
    uchar *d_img1, *d_img2, *d_bin1, *d_bin2, *d_xor, *d_display, *d_evidence;
    hipMalloc(&d_img1, total_size);
    hipMalloc(&d_img2, total_size);
    hipMalloc(&d_bin1, total_size);
    hipMalloc(&d_bin2, total_size);
    hipMalloc(&d_xor, total_size);
    hipMalloc(&d_display, total_size_color);
    hipMalloc(&d_evidence, total_size_color);
    
    hipMemcpy(d_img1, img1.data, total_size, hipMemcpyHostToDevice);
    hipMemcpy(d_img2, img2.data, total_size, hipMemcpyHostToDevice);
    hipMemcpy(d_display, display.data, total_size_color, hipMemcpyHostToDevice);

    auto start_time = steady_clock::now();
    
    binarization_kernel<<<nBlocks, NumThreadsPerBlock>>>(d_img1, d_bin1, img1.rows, img1.cols, th);
    binarization_kernel<<<nBlocks, NumThreadsPerBlock>>>(d_img2, d_bin2, img2.rows, img2.cols, th);
    xor_kernel<<<nBlocks, NumThreadsPerBlock>>>(d_bin1, d_bin2, d_xor, img1.rows, img1.cols);
    mark_differences_kernel<<<nBlocks, NumThreadsPerBlock>>>(d_xor, d_display, d_evidence, img1.rows, img1.cols);

    auto end_time = steady_clock::now();
    duration<double> elapsed_seconds = end_time - start_time;
    cout << "EXECUTION TIME: " << fixed << setprecision(6) << elapsed_seconds.count() << endl << endl;

    Mat xor_result(img1.size(), CV_8UC1);
    Mat evidence(img1.size(), CV_8UC3);
    hipMemcpy(xor_result.data, d_xor, total_size, hipMemcpyDeviceToHost);
    hipMemcpy(evidence.data, d_evidence, total_size_color, hipMemcpyDeviceToHost);
    
    if (imwrite("outputs/evidence_result_parallel.jpg", evidence) && imwrite("outputs/XOR_parallel.jpg", xor_result)) {
        cout << "Images saved!\n";
    } else {
        cerr << "Error saving images!\n";
    }

    hipFree(d_img1); 
    hipFree(d_img2); 
    hipFree(d_bin1);
    hipFree(d_bin2); 
    hipFree(d_xor); 
    hipFree(d_display); 
    hipFree(d_evidence);
    return 0;
}


/********** CUDA KERNELS **********/

__global__ void binarization_kernel(uchar* img, uchar* bin_result, int rows, int cols, int th) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (y < rows && x < cols) {
        bin_result[y * cols + x] = (img[y * cols + x] >= th) ? 255 : 0;
    }
}

__global__ void xor_kernel(uchar* img1, uchar* img2, uchar* xor_result, int rows, int cols) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (y < rows && x < cols) {
        int idx = y * cols + x;
        xor_result[idx] = img1[idx] ^ img2[idx];
    }
}

__global__ void mark_differences_kernel(uchar* xor_result, uchar* color_src, uchar* evidence, int rows, int cols) {
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;
    if (y < rows && x < cols) {
        int idx = y * cols + x;
        if (xor_result[idx] > 0) {
            evidence[3 * idx] = color_src[3 * idx];
            evidence[3 * idx + 1] = color_src[3 * idx + 1];
            evidence[3 * idx + 2] = color_src[3 * idx + 2];
        }
    }
}
